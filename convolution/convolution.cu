#include "hip/hip_runtime.h"
/*
 * Original source from nvidia cuda SDK 2.0
 * Modified by S. James Lee (sjames@evl.uic.edi)
 * 2008.12.05
 */

/*
 * This sample implements a separable convolution filter 
 * of a 2D signal with a gaussian kernel.
 */


#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cutil.h>

////////////////////////////////////////////////////////////////////////////////
// Image file loader (RAW format)
////////////////////////////////////////////////////////////////////////////////
bool loadRawImage(char* filename, int w, int h, float* r, float* g, float* b)
{
	FILE *imageFile;
	imageFile = fopen(filename, "r");
    
	if (imageFile == NULL) 
	{
		printf("Cannot find texture file in data directory: %s\n", filename);
		return false;
	}
	else
	{	    
		for (int i = 0; i < h*w; i+=1)
		{
			r[i]	= fgetc(imageFile);
			g[i]	= fgetc(imageFile);
			b[i]	= fgetc(imageFile);
		}            
		
		fclose(imageFile);
		return true;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Image file writer (RAW format)
////////////////////////////////////////////////////////////////////////////////
bool writeRawImage(char* filename, int w, int h, float* r, float* g, float* b)
{
	FILE *imageFile;
	imageFile = fopen(filename, "wb");
    
	if (imageFile == NULL) 
	{
		printf("Cannot write texture file: %s\n", filename);
		return false;
	}
	else
	{	  
		for (int i = 0; i < h*w; i+=1)
		{
			fputc((int)(r[i]), imageFile);
			fputc((int)(g[i]), imageFile);
			fputc((int)(b[i]), imageFile);
		}
		            
		fclose(imageFile);
		return true;
	}
    
}


////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Round a / b to nearest lower integer value
int iDivDown(int a, int b){
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b){
    return a - a % b;
}

// Convert float r,g,b to int type
__device__ int rgbToint(float r, float g, float b, float a){
    return
        ((int)(a * 255.0f) << 24) |
        ((int)(b * 255.0f) << 16) |
        ((int)(g * 255.0f) <<  8) |
        ((int)(r * 255.0f) <<  0);
}

////////////////////////////////////////////////////////////////////////////////
// GPU convolution
////////////////////////////////////////////////////////////////////////////////
//Global macro, controlling innermost convolution loop unrolling
#define UNROLL_INNER
#include <convolution_kernel.cu>


////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//Image width should be aligned to maximum coalesced read/write size
//for best global memory performance in both row and column filter.
const int KERNEL_SIZE = KERNEL_W * sizeof(float);


//Carry out dummy calculations before main computation loop
//in order to "warm up" the hardhare/driver
//#define WARMUP

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    
    float *h_Kernel;
	float *h_DataR, *h_DataG, *h_DataB, *h_ResultR, *h_ResultG, *h_ResultB;
    float *d_DataA, *d_DataB;

    double gpuTime, runTime, singleRunTime;

    int i, dw, dh, data_size, repeat;
	dw = dh = 1024;
	repeat = 10;
	
    unsigned int hTimer;

    CUT_DEVICE_INIT(argc, argv);
    CUT_SAFE_CALL(cutCreateTimer(&hTimer));

	// check arg: image resolution
	char *iFilename = "../../../../hubble/hubble1kby1k.raw";
	char *oFilename = "hubble1kby1k_out.raw";
	cutGetCmdLineArgumenti(argc, (const char**) argv, "i", &dw);
	switch (dw)
    {
    case 1024:
    	iFilename = "../../../../hubble/hubble1kby1k.raw";
    	oFilename = "hubble1kby1k_out.raw";
    	dh = dw;
    	break;
    case 2048:
    	iFilename = "../../../../hubble/hubble2kby2k.raw";
    	oFilename = "hubble2kby2k_out.raw";
    	dh = dw;
    	break;
    case 4096:
    	iFilename = "../../../../hubble/hubble4kby4k.raw";
    	oFilename = "hubble4kby4k_out.raw";
    	dh = dw;
    	break;
    default:
    	dh = dw = 1024;
    	printf("use image resoluiton one of 1024, 2048, 4096...\n");
    	printf("will use 1024x1024 as default resolution this time.\n");
    	break;
    }	
	data_size = dw * dh * sizeof(int);
	
	// total iteration number for mean value
	cutGetCmdLineArgumenti(argc, (const char**) argv, "n", &repeat);
	
    printf("Initializing data...\n");
    h_Kernel    = (float *)malloc(KERNEL_SIZE);
    
    h_DataR		= (float *)malloc(data_size);
    h_DataG		= (float *)malloc(data_size);
    h_DataB		= (float *)malloc(data_size);
    h_ResultR	= (float *)malloc(data_size);
    h_ResultG	= (float *)malloc(data_size);
    h_ResultB	= (float *)malloc(data_size);
    
    CUDA_SAFE_CALL( hipMalloc( (void **)&d_DataA, data_size) );
    CUDA_SAFE_CALL( hipMalloc( (void **)&d_DataB, data_size) );

	// initialize kernel
    float kernelSum = 0;
    for(i = 0; i < KERNEL_W; i++){
        float dist = (float)(i - KERNEL_RADIUS) / (float)KERNEL_RADIUS;
        h_Kernel[i] = expf(- dist * dist / 2);
        kernelSum += h_Kernel[i];
    }
    for(i = 0; i < KERNEL_W; i++)
        h_Kernel[i] /= kernelSum;

    if (!loadRawImage(iFilename, dw, dh, h_DataR, h_DataG, h_DataB) )
    {
    	printf("File not found. random image generator will be used...\n");
        	
      	srand(2007);
       	for(i = 0; i < dw * dh; i++)
       	{
       	    h_DataR[i] = (float)rand() / (float)RAND_MAX * 255.0f;
       	    h_DataG[i] = (float)rand() / (float)RAND_MAX * 255.0f;
       	    h_DataB[i] = (float)rand() / (float)RAND_MAX * 255.0f;
       	}       	
    }
    
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, KERNEL_SIZE) );

	//
	//dim3 blocks(32, 32);
	//dim3 grids(dw/16, dh/16);
	dim3 blocks(TILE_W, TILE_H);
	dim3 grids(dw/TILE_W, dh/TILE_H);
	
	//writeRawImage(oFilename, dw, dh, h_DataR, h_DataG, h_DataB);
/*
#ifdef WARMUP
    printf("Warm up ");

	for (i=0; i<40; i++)
	{    
    	// red channel
    	CUDA_SAFE_CALL( hipMemcpy(d_DataA, h_DataR, data_size, hipMemcpyHostToDevice) );
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	convolutionGPU<<<grids, blocks>>>(
            d_DataB,
            d_DataA,
            dw,
            dh
    	);
    	CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );

    	// green channel
    	CUDA_SAFE_CALL( hipMemcpy(d_DataA, h_DataG, data_size, hipMemcpyHostToDevice) );
       	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	convolutionGPU<<<grids, blocks>>>(
            d_DataB,
            d_DataA,
            dw,
            dh
    	);
    	CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );

    	// blue channel
    	CUDA_SAFE_CALL( hipMemcpy(d_DataA, h_DataB, data_size, hipMemcpyHostToDevice) );
      	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	convolutionGPU<<<grids, blocks>>>(
            d_DataB,
            d_DataA,
            dw,
            dh
    	);
    	CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );

    	printf(".");
	}
	printf("\n");
#endif
*/
	printf("=============================================================\n");
	printf(" CUDA Convolution: Image Resolution %i x %i\n", dw, dh);
	printf("=============================================================\n");

	runTime = 0;	
	for (i = 0; i < repeat; i++)
	{
    	// red channel
    	CUDA_SAFE_CALL( hipMemcpy(d_DataA, h_DataR, data_size, hipMemcpyHostToDevice) );
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	CUT_SAFE_CALL( cutResetTimer(hTimer) );
    	CUT_SAFE_CALL( cutStartTimer(hTimer) );
    	convolutionRowGPU<<<grids, blocks>>>(
            d_DataB,
            d_DataA,
            dw,
            dh
    	);
    	CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    	convolutionColGPU<<<grids, blocks>>>(
            d_DataA,
            d_DataB,
            dw,
            dh
    	);
		CUT_CHECK_ERROR("convolutionColGPU() execution failed\n");
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	CUT_SAFE_CALL(cutStopTimer(hTimer));
    	gpuTime = cutGetTimerValue(hTimer);
    	runTime += gpuTime;
    	singleRunTime = gpuTime;

		// read back GPU result
		if (i == (repeat -1))
    		CUDA_SAFE_CALL( hipMemcpy(h_ResultR, d_DataA, data_size, hipMemcpyDeviceToHost) );
    	
    	// green channel
    	CUDA_SAFE_CALL( hipMemcpy(d_DataA, h_DataG, data_size, hipMemcpyHostToDevice) );
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	CUT_SAFE_CALL( cutResetTimer(hTimer) );
    	CUT_SAFE_CALL( cutStartTimer(hTimer) );
    	convolutionRowGPU<<<grids, blocks>>>(
            d_DataB,
            d_DataA,
            dw,
            dh
    	);
    	CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    	convolutionColGPU<<<grids, blocks>>>(
            d_DataA,
            d_DataB,
            dw,
            dh
    	);
		CUT_CHECK_ERROR("convolutionColGPU() execution failed\n");
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	CUT_SAFE_CALL(cutStopTimer(hTimer));
    	gpuTime = cutGetTimerValue(hTimer);
    	runTime += gpuTime;
    	singleRunTime += gpuTime;

		// read back GPU result
		if (i == (repeat -1))
    		CUDA_SAFE_CALL( hipMemcpy(h_ResultG, d_DataA, data_size, hipMemcpyDeviceToHost) );

    	// blue channel
    	CUDA_SAFE_CALL( hipMemcpy(d_DataA, h_DataB, data_size, hipMemcpyHostToDevice) );
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	CUT_SAFE_CALL( cutResetTimer(hTimer) );
    	CUT_SAFE_CALL( cutStartTimer(hTimer) );
    	convolutionRowGPU<<<grids, blocks>>>(
            d_DataB,
            d_DataA,
            dw,
            dh
    	);
    	CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    	convolutionColGPU<<<grids, blocks>>>(
            d_DataA,
            d_DataB,
            dw,
            dh
    	);
		CUT_CHECK_ERROR("convolutionColGPU() execution failed\n");
    	CUDA_SAFE_CALL( hipDeviceSynchronize() );
    	CUT_SAFE_CALL(cutStopTimer(hTimer));
    	gpuTime = cutGetTimerValue(hTimer);
    	runTime += gpuTime;
    	singleRunTime += gpuTime;
    	printf("%ith GPU convolution time : %f msec\n", i, singleRunTime);

		// read back GPU result
		if (i == (repeat -1))
    		CUDA_SAFE_CALL( hipMemcpy(h_ResultB, d_DataA, data_size, hipMemcpyDeviceToHost) );

	}

	printf("=============================================================\n");
	printf(" Convolution Time: %f msecs (mean of %i run)\n", runTime/ repeat, repeat);
	printf("=============================================================\n\n");
	
	// write result image
	writeRawImage(oFilename, dw, dh, h_ResultR, h_ResultG, h_ResultB);

    printf("Shutting down...\n");
    CUDA_SAFE_CALL( hipFree(d_DataB) );
    CUDA_SAFE_CALL( hipFree(d_DataA) );
    
    free(h_ResultB);
    free(h_ResultG);
    free(h_ResultR);
    free(h_DataB);
    free(h_DataG);
    free(h_DataR);
    free(h_Kernel);

    CUT_SAFE_CALL(cutDeleteTimer(hTimer));
}
