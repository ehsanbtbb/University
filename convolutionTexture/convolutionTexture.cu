#include "hip/hip_runtime.h"
/*
 * Original source from nvidia cuda SDK 2.0
 * Modified by S. James Lee (sjames@evl.uic.edi)
 * 2008.12.05
 */


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cutil.h>


////////////////////////////////////////////////////////////////////////////////
// Image file loader (RAW format)
////////////////////////////////////////////////////////////////////////////////
bool loadRawImage(char* filename, int w, int h, float* r, float* g, float* b)
{
	FILE *imageFile;
	imageFile = fopen(filename, "r");
    
	if (imageFile == NULL) 
	{
		printf("Cannot find texture file in data directory: %s\n", filename);
		return false;
	}
	else
	{	    
		for (int i = 0; i < h*w; i+=1)
		{
			r[i]	= fgetc(imageFile);
			g[i]	= fgetc(imageFile);
			b[i]	= fgetc(imageFile);
		}            
		
		fclose(imageFile);
		return true;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Image file writer (RAW format)
////////////////////////////////////////////////////////////////////////////////
bool writeRawImage(char* filename, int w, int h, float* r, float* g, float* b)
{
	FILE *imageFile;
	imageFile = fopen(filename, "wb");
    
	if (imageFile == NULL) 
	{
		printf("Cannot write texture file: %s\n", filename);
		return false;
	}
	else
	{	  
		for (int i = 0; i < h*w; i+=1)
		{
			fputc((int)(r[i]), imageFile);
			fputc((int)(g[i]), imageFile);
			fputc((int)(b[i]), imageFile);
		}
		            
		fclose(imageFile);
		return true;
	}
    
}

////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////
//Round a / b to nearest higher integer value
int iDivUp(int a, int b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b){
    return (a % b != 0) ?  (a - a % b + b) : a;
}


////////////////////////////////////////////////////////////////////////////////
// GPU convolution
////////////////////////////////////////////////////////////////////////////////
#define UNROLL_INNER
#include "convolutionTexture_kernel.cu"



////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//const int	dw = iAlignUp(4096, 16);
//const int	dh = 4096;

const int	KERNEL_SIZE = KERNEL_W * sizeof(float);
//const int	data_size = dw   * dh   * sizeof(float);


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv){
    
    float *h_Kernel;
    float *h_DataR, *h_DataG, *h_DataB;
	float *h_ResultR, *h_ResultG, *h_ResultB;
	
    hipArray *a_Data;

    hipChannelFormatDesc floatTex = hipCreateChannelDesc<float>();

    float *d_Result;

    double gpuTime, runTime;

    int i, dw, dh, data_size, repeat;
    dw = dh = 1024;
    repeat = 10;

    unsigned int hTimer;


    CUT_DEVICE_INIT(argc, argv);
    CUT_SAFE_CALL( cutCreateTimer(&hTimer) );
	
	// check arg: image resolution
	char *iFilename = "../../../../hubble/hubble1kby1k.raw";
	char *oFilename = "hubble1kby1k_out.raw";
	cutGetCmdLineArgumenti(argc, (const char**) argv, "i", &dw);
	switch (dw)
    {
    case 1024:
    	iFilename = "../../../../hubble/hubble1kby1k.raw";
    	oFilename = "hubble1kby1k_out.raw";
    	dh = dw;
    	break;
    case 2048:
    	iFilename = "../../../../hubble/hubble2kby2k.raw";
    	oFilename = "hubble2kby2k_out.raw";
    	dh = dw;
    	break;
    case 4096:
    	iFilename = "../../../../hubble/hubble4kby4k.raw";
    	oFilename = "hubble4kby4k_out.raw";
    	dh = dw;
    	break;
    default:
    	dh = dw = 1024;
    	printf("use image resoluiton one of 1024, 2048, 4096...\n");
    	printf("will use 1024x1024 as default resolution this time.\n");
    	break;
    }	
	data_size = dw * dh * sizeof(int);
	
	// total iteration number for mean value
	cutGetCmdLineArgumenti(argc, (const char**) argv, "n", &repeat);
		
    printf("Initializing data...\n");
        h_Kernel    = (float *)malloc(KERNEL_SIZE);
        h_DataR     = (float *)malloc(data_size);
        h_DataG     = (float *)malloc(data_size);
        h_DataB     = (float *)malloc(data_size);
        h_ResultR = (float *)malloc(data_size);
        h_ResultG = (float *)malloc(data_size);
        h_ResultB = (float *)malloc(data_size);
        CUDA_SAFE_CALL( hipMallocArray(&a_Data, &floatTex, dw, dh) );
        CUDA_SAFE_CALL( hipMalloc((void **)&d_Result, data_size) );

        float kernelSum = 0;
        for(i = 0; i < KERNEL_W; i++){
            float dist = (float)(i - KERNEL_RADIUS) / (float)KERNEL_RADIUS;
            h_Kernel[i] = expf(- dist * dist / 2);
            kernelSum += h_Kernel[i];
        }
        for(i = 0; i < KERNEL_W; i++)
            h_Kernel[i] /= kernelSum;
        
    	if (!loadRawImage(iFilename, dw, dh, h_DataR, h_DataG, h_DataB) )
    	{
    		printf("File not found. random image generator will be used...\n");
        	
      		srand(2007);
       		for(i = 0; i < dw * dh; i++)
       		{
       	    	h_DataR[i] = (float)rand() / (float)RAND_MAX * 255.0f;
       	    	h_DataG[i] = (float)rand() / (float)RAND_MAX * 255.0f;
       	    	h_DataB[i] = (float)rand() / (float)RAND_MAX * 255.0f;
       		}       	
    	}

        CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), h_Kernel, KERNEL_SIZE) );
        CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, h_DataR, data_size, hipMemcpyHostToDevice) );
        CUDA_SAFE_CALL( hipBindTextureToArray(texData, a_Data) );

        //Block width should be a multiple of maximum coalesced write size
        //for coalesced memory writes in convolutionRowGPU() and convolutionColumnGPU()
        dim3 threadBlock(16, 12);
        dim3 blockGrid(iDivUp(dw, threadBlock.x), iDivUp(dh, threadBlock.y));


///////////////////////
// Warm up
printf("warming up");
for (i = 0; i < 1; i++)
{
    printf(".");
    
    // red
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, h_DataR, data_size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    convolutionRowGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    
    //While CUDA kernels can't  directly write to textures, this copy is inevitable
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, d_Result, data_size, hipMemcpyDeviceToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    convolutionColumnGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionColumnGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
	// green
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, h_DataG, data_size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    convolutionRowGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    
    //While CUDA kernels can't  directly write to textures, this copy is inevitable
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, d_Result, data_size, hipMemcpyDeviceToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    convolutionColumnGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionColumnGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
	// blue
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, h_DataB, data_size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    convolutionRowGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    
    //While CUDA kernels can't  directly write to textures, this copy is inevitable
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, d_Result, data_size, hipMemcpyDeviceToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    convolutionColumnGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionColumnGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
}
printf("\n");

//////////////////////////////////
	printf("=============================================================\n");
	printf(" CUDA Convolution: Image Resolution %i x %i\n", dw, dh);
	printf("=============================================================\n");
		
runTime = 0;
for (i = 0; i < repeat; i++)
{
    // red
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, h_DataR, data_size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    convolutionRowGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime = cutGetTimerValue(hTimer);
    
    //While CUDA kernels can't  directly write to textures, this copy is inevitable
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, d_Result, data_size, hipMemcpyDeviceToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    convolutionColumnGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionColumnGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime += cutGetTimerValue(hTimer);
	
	if (i == (repeat-1))
		CUDA_SAFE_CALL( hipMemcpy(h_ResultR, d_Result, data_size, hipMemcpyDeviceToHost) );
	
	// green
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, h_DataG, data_size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    convolutionRowGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime += cutGetTimerValue(hTimer);
    
    //While CUDA kernels can't  directly write to textures, this copy is inevitable
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, d_Result, data_size, hipMemcpyDeviceToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    convolutionColumnGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionColumnGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime += cutGetTimerValue(hTimer);
	
	if (i == (repeat-1))
		CUDA_SAFE_CALL( hipMemcpy(h_ResultG, d_Result, data_size, hipMemcpyDeviceToHost) );
	
	// blue
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, h_DataB, data_size, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    convolutionRowGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionRowGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime += cutGetTimerValue(hTimer);
    
    //While CUDA kernels can't  directly write to textures, this copy is inevitable
    CUDA_SAFE_CALL( hipMemcpyToArray(a_Data, 0, 0, d_Result, data_size, hipMemcpyDeviceToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    CUT_SAFE_CALL( cutResetTimer(hTimer) );
    CUT_SAFE_CALL( cutStartTimer(hTimer) );
    convolutionColumnGPU<<<blockGrid, threadBlock>>>( d_Result, dw, dh );
    CUT_CHECK_ERROR("convolutionColumnGPU() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUT_SAFE_CALL( cutStopTimer(hTimer) );
    gpuTime += cutGetTimerValue(hTimer);
	runTime += gpuTime;
	printf("%ith GPU convolution time : %f msec\n", i, gpuTime);
	
	if (i == (repeat-1))
		CUDA_SAFE_CALL( hipMemcpy(h_ResultB, d_Result, data_size, hipMemcpyDeviceToHost) );

}

	printf("=============================================================\n");
	printf(" Convolution Time: %f msecs (mean of %i run)\n", runTime/ repeat, repeat);
	printf("=============================================================\n\n");
	
	// write result image
	writeRawImage(oFilename, dw, dh, h_ResultR, h_ResultG, h_ResultB);
		
    printf("Shutting down...\n");
        CUDA_SAFE_CALL( hipUnbindTexture(texData) );
        CUDA_SAFE_CALL( hipFree(d_Result)   );
        CUDA_SAFE_CALL( hipFreeArray(a_Data)   );
        free(h_ResultB);
        free(h_ResultG);
        free(h_ResultR);
        free(h_DataB);
        free(h_DataG);
        free(h_DataR);
        free(h_Kernel);
        
    //CUT_EXIT(argc, argv);
}
